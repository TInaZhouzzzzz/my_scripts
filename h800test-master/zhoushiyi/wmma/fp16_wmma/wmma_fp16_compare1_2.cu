#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
using namespace std;

#include <sys/time.h>
#include <unistd.h>
#include <stdlib.h>
#include <cstdlib>


union FP32
{
    unsigned int i;
    float f;
};


union FP16
{
    unsigned short int i;
    __half f;
};

__global__ void test(float* dst, __half* a, __half* b, float* c){
  asm volatile(
    "ld.param.u64    %rd1, [_Z4testPfP6__halfS1_S__param_0];\n\t"
    ".reg .b32 a<8>, b<8>, c<8>,d<8>;\n\t"
    "wmma.load.a.sync.aligned.m16n16k16.global.row.f16 {a0, a1, a2, a3, a4, a5, a6, a7}, [%1];\n\t"
    "wmma.load.b.sync.aligned.m16n16k16.global.col.f16 {b0, b1, b2, b3, b4, b5, b6, b7}, [%2];\n\t"
    "wmma.load.c.sync.aligned.m16n16k16.global.row.f32 {c0, c1, c2, c3, c4, c5, c6, c7}, [%3];\n\t"
    "wmma.mma.sync.aligned.m16n16k16.row.col.f32.f32 {d0,d1,d2,d3,d4,d5,d6,d7},  {a0, a1, a2, a3, a4, a5, a6, a7}, {b0, b1, b2, b3, b4, b5, b6, b7}, {c0, c1, c2, c3, c4, c5, c6, c7};\n\t"
    "wmma.store.d.sync.aligned.m16n16k16.global.row.f32 [%0], {d0,d1,d2,d3,d4,d5,d6,d7};" : "=l"(dst): "l"(a), "l"(b), "l"(c));
}

void InitOne(__half* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 1.0;
  }
}

void InitZero(__half* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}


void InitZero_float(float* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}

void show(float * a, const int n) {
  std::cout << std::endl;
  for ( int i=0; i<n; i++){ 
    std::cout<<a[i] << std::endl;
  }
  std::cout << std::endl;
}


int main(int argc, char** argv){
  int size = 256;
  __half* host_a=(__half*)malloc(sizeof(__half) * size);
  __half* host_b=(__half*)malloc(sizeof(__half) * size);
  float* host_c=(float*)malloc(sizeof(float) * size);
  float* host_d=(float*)malloc(sizeof(float) * size);
  __half* device_a=NULL;
  __half* device_b=NULL;
  float* device_c=NULL;
  float* device_d=NULL;
  hipMalloc((void**)(&device_a), sizeof(__half) * size);
  hipMalloc((void**)(&device_b), sizeof(__half) * size);
  hipMalloc((void**)(&device_c), sizeof(float) * size);
  hipMalloc((void**)(&device_d), sizeof(float) * size);
  InitZero(host_a, size);
  InitOne(host_b, size);
  InitZero_float(host_c, size);
  InitZero_float(host_d, size);

  FP16 fp16;
  fp16.i = 0x7000; host_a[0]=fp16.f;
  fp16.i = 0x0c00; host_a[1]=fp16.f;
  fp16.i = 0x0c00; host_a[2]=fp16.f;
  fp16.i = 0x0c00; host_a[3]=fp16.f;
  fp16.i = 0x0c00; host_a[4]=fp16.f;
  fp16.i = 0x0c00; host_a[5]=fp16.f;
  fp16.i = 0x0c00; host_a[6]=fp16.f;
  fp16.i = 0x0c00; host_a[7]=fp16.f;

  hipMemcpy((void*)device_a, (void*)host_a, sizeof(__half)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_b, (void*)host_b, sizeof(__half)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_c, (void*)host_c, sizeof(float)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_d, (void*)host_d, sizeof(float)* size, hipMemcpyHostToDevice);

  test<<<1,32>>>(device_d, device_a, device_b, device_c);
  hipDeviceSynchronize();

  hipMemcpy((void*)host_d, (void*)device_d, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  FP32 fp32;
  fp32.f=host_d[0];
//std::cout<< host_d[0] << std::endl;
  std::cout<< hex << fp32.i << std::endl;
//show(host_d, size);
}
