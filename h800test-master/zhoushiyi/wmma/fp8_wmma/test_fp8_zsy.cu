#include <iostream>
#include <cstdlib>
using namespace std;

#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>


union FP32
{
    unsigned int i;
    float f;
};

__global__ void test(float* d, float* a, float* b){
  float d_array[4];
  for (int i = 0; i < 4; ++i) {
    d_array[i] = d[i];
  }

  asm volatile(
      ".reg .b32 ra<4>, rd<4>;\n\t"
      "wgmma.fence.sync.aligned;\n\t"
      "wgmma.mma_async.sync.aligned.m64n8k32.f32.e4m3.e5m2 {%0, %1, %2, %3}, %4, %5, 0, -1, -1;\n\t"
      "wgmma.commit_group.sync.aligned;\n\t"
      "wgmma.wait_group.sync.aligned 0;\n\t" 
      : "+f"(d_array[0]), "+f"(d_array[1]), "+f"(d_array[2]), "+f"(d_array[3])
      : "l"(a),"l"(b)
      );
  for (int i = 0; i < 4; ++i) {
    d[i] = d_array[i];
  }
}

void InitOne(float * a, const int n) {
  FP32 fp32;
  fp32.i = 0x38383838;
  for ( int i = 0; i < n; i++ ) {
	  a[i] = fp32.f;
  }
}
void InitZero(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}

void show(float * a, const int n) {
  for ( int i=0; i<n; i++){ 
    std::cout << a[i] << std::endl;
  }
  std::cout << std::endl;
}

int main(int argc, char** argv){
  int size = 512;
  float* host_a=(float*)malloc(sizeof(float) * size);
  float* host_b=(float*)malloc(sizeof(float) * size);
  float* host_d=(float*)malloc(sizeof(float) * size);
  float* device_a=NULL;
  float* device_b=NULL;
  float* device_d=NULL;
  hipMalloc((void**)(&device_a), sizeof(float) * size);
  hipMalloc((void**)(&device_b), sizeof(float) * size);
  hipMalloc((void**)(&device_d), sizeof(float) * size);
  for(int i=0;i<size;i++){
    host_a[i] =0.0;
    host_d[i] = 0.0;
  }
  InitOne(host_b, size);

  FP32 fp32;
  fp32.i = 0x70400000;  host_a[0]=fp32.f;

  hipMemcpy((void*)device_a, (void*)host_a, sizeof(float)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_b, (void*)host_b, sizeof(float)* size, hipMemcpyHostToDevice);

  test<<<4,128>>>(device_d, device_a, device_b);
  hipDeviceSynchronize();
  hipMemcpy((void*)host_d, (void*)device_d, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  fp32.f=host_d[0];
  std::cout<< hex << fp32.i << std::endl;
//show(host_d, size);

}

