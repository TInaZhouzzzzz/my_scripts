#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <stdio.h>
using namespace std;

#include <sys/time.h>
#include <unistd.h>
#include <stdlib.h>
#include <cstdlib>


union FP32
{
    unsigned int i;
    float f;
};


union BF16
{
    unsigned short int i;
    __hip_bfloat16 f;
};

__global__ void test(float* dst, __hip_bfloat16* a, __hip_bfloat16* b, float* c){
  asm volatile(
//  "ld.param.u64    %rd1, [_Z4testPfP6__halfS1_S__param_0];\n\t"
    "ld.param.u64 	%rd1, [_Z4testPfP13__nv_bfloat16S1_S__param_0];\n\t"
    ".reg .b32 a<4>, b<4>, c<8>,d<8>;\n\t"
    "wmma.load.a.sync.aligned.m16n16k16.global.row.bf16 {a0, a1, a2, a3}, [%1];\n\t"
    "wmma.load.b.sync.aligned.m16n16k16.global.col.bf16 {b0, b1, b2, b3}, [%2];\n\t"
    "wmma.load.c.sync.aligned.m16n16k16.global.row.f32 {c0, c1, c2, c3, c4, c5, c6, c7}, [%3];\n\t"
    "wmma.mma.sync.aligned.m16n16k16.row.col.f32.bf16.bf16.f32 {d0,d1,d2,d3,d4,d5,d6,d7},  {a0, a1, a2, a3}, {b0, b1, b2, b3}, {c0, c1, c2, c3, c4, c5, c6, c7};\n\t"
    "wmma.store.d.sync.aligned.m16n16k16.global.row.f32 [%0], {d0,d1,d2,d3,d4,d5,d6,d7};" : "=l"(dst): "l"(a), "l"(b), "l"(c));
}

void InitOne(__hip_bfloat16* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 1.0;
  }
}

void InitZero(__hip_bfloat16* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}


void InitZero_float(float* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}

void show(float * a, const int n) {
  std::cout << std::endl;
  for ( int i=0; i<n; i++){ 
    std::cout<<a[i] << std::endl;
  }
  std::cout << std::endl;
}


int main(int argc, char** argv){
  int size = 256;
  __hip_bfloat16* host_a=(__hip_bfloat16*)malloc(sizeof(__hip_bfloat16) * size);
  __hip_bfloat16* host_b=(__hip_bfloat16*)malloc(sizeof(__hip_bfloat16) * size);
  float* host_c=(float*)malloc(sizeof(float) * size);
  float* host_d=(float*)malloc(sizeof(float) * size);
  __hip_bfloat16* device_a=NULL;
  __hip_bfloat16* device_b=NULL;
  float* device_c=NULL;
  float* device_d=NULL;
  hipMalloc((void**)(&device_a), sizeof(__hip_bfloat16) * size);
  hipMalloc((void**)(&device_b), sizeof(__hip_bfloat16) * size);
  hipMalloc((void**)(&device_c), sizeof(float) * size);
  hipMalloc((void**)(&device_d), sizeof(float) * size);
  InitZero(host_a, size);
  InitOne(host_b, size);
  InitZero_float(host_c, size);
  InitZero_float(host_d, size);

  BF16 bf16;
  bf16.i = 0x3f80; host_a[0]=bf16.f;
  bf16.i = 0x3f80; host_a[1]=bf16.f;
  bf16.i = 0x3f80; host_a[2]=bf16.f;
  bf16.i = 0x3f80; host_a[3]=bf16.f;
  bf16.i = 0x3f80; host_a[4]=bf16.f;
  bf16.i = 0x3f80; host_a[5]=bf16.f;
  bf16.i = 0x3f80; host_a[6]=bf16.f;
  bf16.i = 0x3f80; host_a[7]=bf16.f;


  FP32 fp32;
  fp32.i = 0x4c000000; host_c[0]=fp32.f;

  hipMemcpy((void*)device_a, (void*)host_a, sizeof(__hip_bfloat16)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_b, (void*)host_b, sizeof(__hip_bfloat16)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_c, (void*)host_c, sizeof(float)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_d, (void*)host_d, sizeof(float)* size, hipMemcpyHostToDevice);

  test<<<1,32>>>(device_d, device_a, device_b, device_c);
  hipDeviceSynchronize();

  hipMemcpy((void*)host_d, (void*)device_d, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

//FP32 fp32;
  fp32.f=host_d[0];
//std::cout<< host_d[0] << std::endl;
  std::cout<< hex << fp32.i << std::endl;
//show(host_d, size);
}
