#include <iostream>
#include <cstdlib>
using namespace std;

#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>


union FP32
{
    unsigned int i;
    float f;
};

__global__ void test(float* d, float* a, float* b, float* c){
  asm volatile(
    "ld.param.u64    %rd1, [_Z4testPfS_S_S__param_0];\n\t"
    ".reg .b32 a<4>, b<4>, c<8>,d<8>;\n\t"
    "wmma.load.a.sync.aligned.m16n16k8.global.row.tf32 {a0, a1, a2, a3}, [%1];\n\t"
    "wmma.load.b.sync.aligned.m16n16k8.global.col.tf32 {b0, b1, b2, b3}, [%2];\n\t"
    "wmma.load.c.sync.aligned.m16n16k8.global.row.f32 {c0, c1, c2, c3, c4, c5, c6, c7}, [%3];\n\t"
    "wmma.mma.sync.aligned.m16n16k8.row.col.f32.tf32.tf32.f32 {d0,d1,d2,d3,d4,d5,d6,d7},  {a0, a1, a2, a3}, {b0, b1, b2, b3}, {c0, c1, c2, c3, c4, c5, c6, c7};\n\t"
    "wmma.store.d.sync.aligned.m16n16k8.global.row.f32 [%0], {d0,d1,d2,d3,d4,d5,d6,d7};" : "=l"(d): "l"(a), "l"(b), "l"(c));
}

void InitZero(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}

void InitOne(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
      a[i] = 1.0;
  }
}

void Init(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
      a[i] = 1.0 * float(i);
  }
}

void Init_3f800000(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
      FP32 fp32;
      fp32.i = 0x3f800000;
      a[i] = fp32.f;
  }
}


void show(float * a, const int n) {
  std::cout << std::endl;
  for ( int i=0; i<n; i++){ 
    std::cout<<a[i] << std::endl;
  }
  std::cout << std::endl;
}


int main(int argc, char** argv){
  int size = 256;
  float* host_a=(float*)malloc(sizeof(float) * size/2);
  float* host_b=(float*)malloc(sizeof(float) * size/2);
  float* host_c=(float*)malloc(sizeof(float) * size);
  float* host_d=(float*)malloc(sizeof(float) * size);
  float* device_a=NULL;
  float* device_b=NULL;
  float* device_c=NULL;
  float* device_d=NULL;
  hipMalloc((void**)(&device_a), sizeof(float) * size/2);
  hipMalloc((void**)(&device_b), sizeof(float) * size/2);
  hipMalloc((void**)(&device_c), sizeof(float) * size);
  hipMalloc((void**)(&device_d), sizeof(float) * size);
  InitZero(host_a, size/2);
  Init_3f800000(host_b, size/2);
  InitZero(host_c, size);
  InitZero(host_d, size);
  FP32 fp32;
  fp32.i = 0x3f800000; host_a[0]=fp32.f;
  fp32.i = 0x3f800000; host_a[1]=fp32.f;
  fp32.i = 0x3f800000; host_a[4]=fp32.f;
  fp32.i = 0x3f800000; host_a[5]=fp32.f;

  fp32.i = 0x4c000000; host_c[0]=fp32.f;

  hipMemcpy((void*)device_a, (void*)host_a, sizeof(float)* size/2, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_b, (void*)host_b, sizeof(float)* size/2, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_c, (void*)host_c, sizeof(float)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_d, (void*)host_d, sizeof(float)* size, hipMemcpyHostToDevice);

  test<<<1,32>>>(device_d, device_a, device_b, device_c);
  hipDeviceSynchronize();

  hipMemcpy((void*)host_d, (void*)device_d, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  fp32.f=host_d[0];
//std::cout<< host_d[0] << std::endl;
  std::cout<< hex << fp32.i << std::endl;
//show(host_d, size);
}
