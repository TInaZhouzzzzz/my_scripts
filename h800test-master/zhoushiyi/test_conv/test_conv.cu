#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <iostream>
#include <cstdlib>
using namespace std;
#include "src/helper.h"
#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>
#include <hip/hip_bf16.h>
 
void InitRand(float * a, const int n) {
  float value;
  for ( int i = 0; i < n; i++ ) {
    value = (float)(rand() % 20 - 10) + (float)(rand() % 20 - 10) / 10.0 + (float)(rand() % 20 - 10) / 100.0 + (float)(rand() % 20 - 10) / 1000.0 + (float)(rand() % 20 - 10) / 10000.0 + (float)(rand() % 20 - 10) / 100000.0 + (float)(rand() % 20 - 10) / 1000000.0 + (float)(rand() % 20 - 10) / 10000000.0 + (float)(rand() % 20 - 10) / 100000000.0;
    a[i] = value;
  }
}
 
void InitZero(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
    a[i] = 0.f;
  }
}
 
 
void show(float * a, const int n, const int c, const int h, const int w) {
  for ( int i=0; i<n; i++) {
    for ( int j=0; j<c; j++) {
      for ( int k=0; k<h; k++) {
        for ( int l=0; l<w; l++) {
          // std::cout.width(11);
          std::cout << a[i*c + j*h + k*w + l] << ",";
        }
    std::cout << std::endl;
      }
      std::cout << "next channel" << std::endl;
    }
    std::cout << "next batch" << std::endl;
  }
  std::cout << std::endl;
}
 
int main(int argc, char** argv)
{
    hipdnnHandle_t cudnn;
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnTensorDescriptor_t bias_desc;
 
    //hipdnnConvolutionFwdAlgoPerf_t falgo;
    hipdnnConvolutionFwdAlgo_t algo;
     
    float *d_input = nullptr;
    float *d_output = nullptr;
    float *d_output2 = nullptr;
    float *d_filter = nullptr;
    float *d_bias = nullptr;
    float *input, *output, *output2, *filter, *bias;
 
    int input_n = 64;
    int input_c = 128;
    int input_h = 16;
    int input_w = 16;
     
    int pad_h = 0;
    int pad_w = 0;
 
    // output size
    int output_n = 16;
    int output_c = 256;
    int output_h = 64;
    int output_w = 64;
 
    // kernel size
    int filter_h = 1;
    int filter_w = 1;
 
    // alpha, beta
    float one = 1.f;
    float zero = 0.f;
 
    hipdnnCreate(&cudnn);
 
    /* Create Resources */
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnCreateFilterDescriptor(&filter_desc);
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnCreateTensorDescriptor(&bias_desc);
 
    // Initilziae resources
    hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, input_c, input_h, input_w);
    hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, output_c, input_c, filter_h, filter_w);
    hipdnnSetConvolution2dDescriptor(conv_desc,
                                    pad_h, pad_w,
                                    1, 1,
                                    1, 1,
                                    HIPDNN_CROSS_CORRELATION,
                                    HIPDNN_DATA_FLOAT);
    hipdnnSetConvolutionMathType(conv_desc, HIPDNN_DEFAULT_MATH);  //zsy
    hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, filter_desc, &output_n, &output_c, &output_h, &output_w);
    hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, output_n, output_c, output_h, output_w);
    hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, output_c, 1, 1);
     
    int weight_size = output_c * input_c * filter_h * filter_w;
    int bias_size = output_c;
 
    std::cout << "input  size: " << input_n << " " << input_c << " " << input_h << " " << input_w << std::endl;
    std::cout << "output size: " << output_n << " " << output_c << " " << output_h << " " << output_w << std::endl;
    std::cout << "kernel size: " << filter_h << " " << filter_w << std::endl;
    std::cout << "padding size " << pad_h << " " << pad_w << std::endl;
     
    // allocate memory space
    input = (float *)malloc(sizeof(float) * input_n * input_c * input_h * input_w);
    filter = (float *)malloc(sizeof(float) * weight_size);
    output = (float *)malloc(sizeof(float) * output_n * output_c * output_h * output_w); 
    bias = (float *)malloc(sizeof(float) * bias_size);
 
    InitRand(input, input_n * input_c * input_h * input_w);
    input[0] = 12.50390625;
    InitRand(filter, weight_size);
    filter[0] = 1.f;
    InitZero(bias, bias_size);
    InitZero(output, output_n * output_c * output_h * output_w);
 
    std::cout << "Finish init input, filter, bias: " << std::endl;
 
    size_t workspace_size = 0;
    size_t temp_size = 0;
    float *d_workspace = nullptr;
     
    // Algorithm used for convolution
    algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    // algo = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
 
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_desc, filter_desc, conv_desc, output_desc, algo, &temp_size);
 
    workspace_size = max(workspace_size, temp_size);
    std::cout << "algorithm: " << algo << std::endl;
    std::cout << "workspace size: " << workspace_size << std::endl;
 
 
    hipMalloc((void**)&d_input,        sizeof(float) * input_n * input_c * input_h * input_w);
    hipMalloc((void**)&d_filter,       sizeof(float) * weight_size);
    hipMalloc((void**)&d_output,       sizeof(float) * output_n * output_c * output_h * output_w);
    hipMalloc((void**)&d_workspace,    sizeof(float) * workspace_size);
    hipMalloc((void**)&d_bias,         sizeof(float) * bias_size);
     
    hipMemcpy(d_input, input, sizeof(float) * input_n * input_c * input_h * input_w, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, sizeof(float) * weight_size, hipMemcpyHostToDevice);
 
    std::cout << "Finish hipMemcpy input and filter " << std::endl;
 
    checkCudnnErrors(hipdnnConvolutionForward(cudnn, &one, input_desc, d_input, filter_desc, d_filter, conv_desc, algo, d_workspace, workspace_size, &zero, output_desc, d_output));
    hipMemcpy(output, d_output, sizeof(float) * output_n * output_c * output_h * output_w, hipMemcpyDeviceToHost);
    std::cout << "Finish compute cudnnConvForward " << std::endl;
 
    //show(output, output_n, output_c, output_h, output_w);
 
    hipFree(d_input);   
    hipFree(d_filter);
    hipFree(d_output);
    hipFree(d_workspace);
    hipFree(d_bias);
 
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyTensorDescriptor(bias_desc);
 
    hipFree(d_input);   
    hipFree(d_filter);
    hipFree(d_output);
    hipFree(d_workspace);
    hipFree(d_bias);
 
    hipdnnDestroy(cudnn);
}
