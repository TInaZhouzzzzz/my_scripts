
#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"
#include <cstdint>

__global__ void wgmma_test1(float *gm_cd, uint64_t *desc) {
  uint64_t a_desc = desc[0], b_desc = desc[1];
  float d_array[4];
  for (int i = 0; i < 4; ++i) {
    d_array[i] = gm_cd[i];
  }
  asm volatile("{\n\t"
               "wgmma.mma_async.sync.aligned.m64n8k16.f32.f16.f16\n\t"
               "{%0, %1, %2, %3}, %4, %5,1,1,1,0,0;\n\t"
               "}\n\t"
               : "+f"(d_array[0]), "+f"(d_array[1]), "+f"(d_array[2]),
                 "+f"(d_array[3])
               : "l"(a_desc), "l"(b_desc)
               :);

  for (int i = 0; i < 4; ++i) {
    gm_cd[i] = d_array[i];
  }
}
