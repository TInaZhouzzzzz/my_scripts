#include "hip/hip_fp16.h"
#include <cstdint>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
using namespace std;

#include <sys/time.h>
#include <unistd.h>
#include <stdlib.h>
#include <cstdlib>


union FP32
{
    unsigned int i;
    float f;
};


union FP16
{
    unsigned short int i;
    __half f;
};

void InitOne(__half* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 1.0;
  }
}

void InitZero(__half* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}


void InitZero_float(float* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}

void show(float * a, const int n) {
  std::cout << std::endl;
  for ( int i=0; i<n; i++){ 
    std::cout<<a[i] << std::endl;
  }
  std::cout << std::endl;
}




__global__ void wgmma_test1(float *gm_cd, __half *a_desc,  __half *b_desc) {
  float d_array[4];
  for (int i = 0; i < 4; ++i) {
    d_array[i] = gm_cd[i];
  }
  printf("hello");
//asm volatile("{\n\t"
//             "wgmma.mma_async.sync.aligned.m64n8k16.f32.f16.f16\n\t"
//             "{%0, %1, %2, %3}, %4, %5,1,1,1,0,0;\n\t"
//             "}\n\t"
//             : "+f"(d_array[0]), "+f"(d_array[1]), "+f"(d_array[2]),
//               "+f"(d_array[3])
//             : "l"(a_desc), "l"(b_desc)
//             :);

  for (int i = 0; i < 4; ++i) {
    gm_cd[i] = d_array[i];
  }
}


int main(int argc, char** argv){
  printf("hello");
  int size = 256;
  __half* host_a=(__half*)malloc(sizeof(__half) * size);
  __half* host_b=(__half*)malloc(sizeof(__half) * size);
//float* host_c=(float*)malloc(sizeof(float) * size);
  float* host_d=(float*)malloc(sizeof(float) * size);
  __half* device_a=NULL;
  __half* device_b=NULL;
//float* device_c=NULL;
  float* device_d=NULL;
  hipMalloc((void**)(&device_a), sizeof(__half) * size);
  hipMalloc((void**)(&device_b), sizeof(__half) * size);
//cudaMalloc((void**)(&device_c), sizeof(float) * size);
  hipMalloc((void**)(&device_d), sizeof(float) * size);
  InitZero(host_a, size);
  InitOne(host_b, size);
//InitZero_float(host_c, size);
  InitZero_float(host_d, size);

  FP16 fp16;
  fp16.i = 0x7000; host_a[0]=fp16.f;
  fp16.i = 0x0c00; host_a[1]=fp16.f;
  fp16.i = 0xffff; host_a[2]=fp16.f;
  fp16.i = 0xffff; host_a[3]=fp16.f;
  fp16.i = 0xffff; host_a[4]=fp16.f;
  fp16.i = 0xffff; host_a[5]=fp16.f;
  fp16.i = 0xffff; host_a[6]=fp16.f;
  fp16.i = 0xffff; host_a[7]=fp16.f;

  hipMemcpy((void*)device_a, (void*)host_a, sizeof(__half)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_b, (void*)host_b, sizeof(__half)* size, hipMemcpyHostToDevice);
//cudaMemcpy((void*)device_c, (void*)host_c, sizeof(float)* size, cudaMemcpyHostToDevice);
  hipMemcpy((void*)device_d, (void*)host_d, sizeof(float)* size, hipMemcpyHostToDevice);

  wgmma_test1<<<1,32>>>(device_d, device_a, device_b);
  hipDeviceSynchronize();

  hipMemcpy((void*)host_d, (void*)device_d, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  FP32 fp32;
  fp32.f=host_d[0];
//std::cout<< host_d[0] << std::endl;
  std::cout<< hex << fp32.i << std::endl;
//show(host_d, size);
}
