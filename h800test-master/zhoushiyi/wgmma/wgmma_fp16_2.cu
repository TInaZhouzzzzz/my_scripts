#include "hip/hip_fp16.h"
#include <cstdint>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
using namespace std;

#include <sys/time.h>
#include <unistd.h>
#include <stdlib.h>
#include <cstdlib>


union FP32
{
    unsigned int i;
    float f;
};


union FP16
{
    unsigned short int i;
    __half f;
};

void InitOne(__half* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 1.0;
  }
}

void InitZero(__half* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}


void InitZero_float(float* a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}

void show(float * a, const int n) {
  std::cout << std::endl;
  for ( int i=0; i<n; i++){ 
    std::cout<<a[i] << std::endl;
  }
  std::cout << std::endl;
}




__global__ void wgmma_test1(float *d, __half *a,  __half *b) {
  asm volatile("{\n\t"
               "ld.param.u64 	%rd2, [_Z11wgmma_test1PfP6__halfS1__param_0];\n\t"
               ".reg .b32 d<4>;\n\t"
               "wgmma.mma_async.sync.aligned.m64n8k16.f32.f16.f16\n\t"
               "{d0, d1, d2, d3}, %1, %2,1,1,1,0,0;\n\t"
               "st.f32 [%0], 1.0;\n\t"
               "}\n\t"
               : "=l"(d) : "l"(a), "l"(b));
}


int main(int argc, char** argv){
  int size = 512;
  __half* host_a=(__half*)malloc(sizeof(__half) * size);
  __half* host_b=(__half*)malloc(sizeof(__half) * size);
//float* host_c=(float*)malloc(sizeof(float) * size);
  float* host_d=(float*)malloc(sizeof(float) * size);
  __half* device_a=NULL;
  __half* device_b=NULL;
//float* device_c=NULL;
  float* device_d=NULL;
  hipMalloc((void**)(&device_a), sizeof(__half) * size);
  hipMalloc((void**)(&device_b), sizeof(__half) * size);
//cudaMalloc((void**)(&device_c), sizeof(float) * size);
  hipMalloc((void**)(&device_d), sizeof(float) * size);
  InitZero(host_a, size);
  InitOne(host_b, size);
//InitZero_float(host_c, size);
  InitZero_float(host_d, size);

  FP16 fp16;
  fp16.i = 0x7000; host_a[0]=fp16.f;
  fp16.i = 0x0c00; host_a[1]=fp16.f;
  fp16.i = 0xffff; host_a[2]=fp16.f;
  fp16.i = 0xffff; host_a[3]=fp16.f;
  fp16.i = 0xffff; host_a[4]=fp16.f;
  fp16.i = 0xffff; host_a[5]=fp16.f;
  fp16.i = 0xffff; host_a[6]=fp16.f;
  fp16.i = 0xffff; host_a[7]=fp16.f;

  hipMemcpy((void*)device_a, (void*)host_a, sizeof(__half)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_b, (void*)host_b, sizeof(__half)* size, hipMemcpyHostToDevice);
//cudaMemcpy((void*)device_c, (void*)host_c, sizeof(float)* size, cudaMemcpyHostToDevice);
  hipMemcpy((void*)device_d, (void*)host_d, sizeof(float)* size, hipMemcpyHostToDevice);

  wgmma_test1<<<1,32>>>(device_d, device_a, device_b);
  hipDeviceSynchronize();

  hipMemcpy((void*)host_d, (void*)device_d, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  FP32 fp32;
  fp32.f=host_d[0];
//std::cout<< host_d[0] << std::endl;
  std::cout<< hex << fp32.i << std::endl;
//show(host_d, size);
}
