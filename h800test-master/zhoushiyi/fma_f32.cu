#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<iostream>
#include <iomanip>      // std::setprecision

#include<hip/hip_fp16.h>
#include "include/half.hpp"
using namespace std;
using half_float::half;
union FP32{
    float f;
    unsigned int i;
};

__global__ void test(float* A, float* B, float* C, float* D){
    asm("fma.rn.f32  %0, %1, %2, %3;" : "=f"(D[0]) : "f"(A[0]) , "f"(B[0]) , "f"(C[0]));
}

int main(int argc, char** argv){
    int size = 1;
    float *dataA = (float*)malloc(sizeof(float) * size);
    float *dataB = (float*)malloc(sizeof(float) * size);
    float *dataC = (float*)malloc(sizeof(float) * size);
    float *dataD = (float*)malloc(sizeof(float) * size);
    float *d_dataA = NULL;
    float *d_dataB = NULL;
    float *d_dataC = NULL;
    float *d_dataD = NULL;
    hipMalloc((void**)&d_dataA, sizeof(float) * size);
    hipMalloc((void**)&d_dataB, sizeof(float) * size);
    hipMalloc((void**)&d_dataC, sizeof(float) * size);
    hipMalloc((void**)&d_dataD, sizeof(float) * size);
    FP32 fp32;

    fp32.i = 0x5d840000;    dataA[size-1] = fp32.f;
    fp32.i = 0xa2300000;    dataB[size-1] = fp32.f;
    fp32.i = 0x01000000;    dataC[size-1] = fp32.f;

//  show(dataA, size);


    for(int i=0;i<size;i++){
        dataD[i] = 0;
    }
    hipMemcpy(d_dataA,dataA,sizeof(float) * size,hipMemcpyHostToDevice);
    hipMemcpy(d_dataB,dataB,sizeof(float) * size,hipMemcpyHostToDevice);
    hipMemcpy(d_dataC,dataC,sizeof(float) * size,hipMemcpyHostToDevice);
    test<<<1, 1>>> (d_dataA, d_dataB, d_dataC, d_dataD);
    hipMemcpy(dataD,d_dataD,sizeof(float) * size, hipMemcpyDeviceToHost);

    fp32.f = dataD[size-1];
    std::cout <<std::hex << fp32.i << std::endl;

    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);
    hipFree(d_dataD);
    free(dataA);
    free(dataB);
    free(dataC); 
    free(dataD); 
    return 0;
}
