#include <iostream>
#include <cstdlib>
using namespace std;

#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>


union FP32
{
    unsigned int i;
    float f;
};

__global__ void test(float* d, float* a, float* b){
  asm volatile(
      "ld.param.u64  %rd1, [_Z4testPfS_S__param_0];\n\t"
      ".reg .b32 f16d<2>;\n\t"
      "wgmma.fence.sync.aligned;\n\t"
      "wgmma.mma_async.sync.aligned.m64n8k32.f16.e4m3.e5m2 {f16d0, f16d1}, %0, %1, 0, 1, 1;\n\t"
      "wgmma.commit_group.sync.aligned;\n\t"
      "wgmma.wait_group.sync.aligned 0;\n\t" 
      "stmatrix.sync.aligned.m8n8.x2.b16 [%0], {f16d0, f16d1};\n\t"
      :"=l"(d): "l"(a),"l"(b)
      );
}

void Initfloat(float * a, const int n) {
  float value;
  for ( int i = 0; i < n; i++ ) {
	value = (float)(rand() % 20 - 10) + (float)(rand() % 20 - 10) / 10.0 + (float)(rand() % 20 - 10) / 100.0 + (float)(rand() % 20 - 10) / 1000.0 + (float)(rand() % 20 - 10) / 10000.0 + (float)(rand() % 20 - 10) / 100000.0 + (float)(rand() % 20 - 10) / 1000000.0 + (float)(rand() % 20 - 10) / 10000000.0 + (float)(rand() % 20 - 10) / 100000000.0;

	a[i] = value;
  }
}

void Initnum(float * a, const int n) {
  FP32 fp32;
  fp32.i = 0x3838;
  for ( int i = 0; i < n; i++ ) {
	  a[i] = fp32.f;
  }
}
void InitZero(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
	  a[i] = 0.0;
  }
}


void show(float * a, const int n) {
  for ( int i=0; i<n; i++){ 
    std::cout << a[i] << std::endl;
  }
  std::cout << std::endl;
}

int main(int argc, char** argv){
  int size = 512;
  float* host_a=(float*)malloc(sizeof(float) * size);
  float* host_b=(float*)malloc(sizeof(float) * size);
  float* host_d=(float*)malloc(sizeof(float) * size);
  float* device_a=NULL;
  float* device_b=NULL;
  float* device_d=NULL;
  hipMalloc((void**)(&device_a), sizeof(float) * size);
  hipMalloc((void**)(&device_b), sizeof(float) * size);
  hipMalloc((void**)(&device_d), sizeof(float) * size);
  FP32 fp32;
  fp32.i = 0x3838;
  for(int i=0;i<size;i++){
    host_a[i] = fp32.f;
  }
  fp32.i = 0x3c3c;
  for(int i=0;i<size;i++){
    host_b[i] = fp32.f;
  }
  fp32.i = 0x3c000000;
  for(int i=0;i<size;i++){
    host_d[i] = fp32.f;
  }
  hipMemcpy((void*)device_a, (void*)host_a, sizeof(float)* size, hipMemcpyHostToDevice);
  hipMemcpy((void*)device_b, (void*)host_b, sizeof(float)* size, hipMemcpyHostToDevice);
  //cudaMemcpy((void*)device_d, (void*)host_d, sizeof(float)* size, cudaMemcpyHostToDevice);

  test<<<4,128>>>(device_d, device_a, device_b);
  hipDeviceSynchronize();
  hipMemcpy((void*)host_d, (void*)device_d, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  fp32.f=host_d[0];
  std::cout<< hex << fp32.i << std::endl;
//show(host_d, size);

}
