#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<iostream>
#include <iomanip>      // std::setprecision

#include<hip/hip_fp16.h>
#include "include/half.hpp"
using namespace std;
using half_float::half;
union FP32{
    float f;
    unsigned int i;
};

union FP16{
    half f;
    unsigned short int i;
};

__global__ void test(float* A, short int* C, int N){
    for(int i=0;i<N;i++){
        //asm("cvt.rn.satfinite.e4m3x2.f32 %0, %1, %2;\n\t":"=h"(C[i]):"f"(A[i]),"f"(B[i]));
//      asm("cvt.rn.relu.satfinite.f16.f32 %0, %1;\n\t":"=h"(C[i]):"f"(A[i]));
        asm("cvt.rn.relu.f16.f32 %0, %1;\n\t":"=h"(C[i]):"f"(A[i]));
    }
}

void Initfloat(float * a, const int n) {
  float value;
  for ( int i = 0; i < n; i++ ) {
    value = (float)(rand() % 20 - 10) + (float)(rand() % 20 - 10) / 10.0 + (float)(rand() % 20 - 10) / 100.0 + (float)(rand() % 20 - 10) / 1000.0 + (float)(rand() % 20 - 10) / 10000.0 + (float)(rand() % 20 - 10) / 100000.0 + (float)(rand() % 20 - 10) / 1000000.0 + (float)(rand() % 20 - 10) / 10000000.0 + (float)(rand() % 20 - 10) / 100000000.0;

    a[i] = value;
  }
}

void InitZero(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
      a[i] = 0.0;
  }
}


void show(float * a, const int n) {
  std::cout << std::endl;
  for ( int i=0; i<n; i++){ 
    std::cout<< std::setprecision(20) << a[i] << std::endl;
  }
  std::cout << std::endl;
}

int main(int argc, char** argv){
    int size = 10;
    float *dataA = (float*)malloc(sizeof(float) * size);
    float *dataB = (float*)malloc(sizeof(float) * size);
    short int *dataC = (short int*)malloc(sizeof(short int) * size);
    float *d_dataA = NULL;
    short int *d_dataC = NULL;
    hipMalloc((void**)&d_dataA, sizeof(float) * size);
    hipMalloc((void**)&d_dataC, sizeof(unsigned short int) * size);
    FP32 fp32;

    Initfloat(dataA, size);
    /* Nan */
    fp32.i = 0x7fffffff;    dataA[size-1] = fp32.f;
    fp32.i = 0xffffffff;    dataA[size-2] = fp32.f;

    /* inf */
    fp32.i = 0x7f800000;    dataA[size-3] = fp32.f;
    fp32.i = 0xff800000;    dataA[size-4] = fp32.f;

    /* 0 */
    fp32.i = 0x00000000;    dataA[size-5] = fp32.f;
    fp32.i = 0x80000000;    dataA[size-6] = fp32.f;

    /* overflow */
    fp32.i = 0x7f7ffeba;    dataA[size-7] = fp32.f;
    fp32.i = 0xff7ffeba;    dataA[size-8] = fp32.f;

    /* overflow */
    fp32.i = 0x43acad91;    dataA[size-9] = fp32.f;
    fp32.i = 0xc3acad91;    dataA[size-10] = fp32.f;


//  /* underflow */
//  fp32.i = 0x36801000;    dataA[count] = fp32.f;

//  show(dataA, size);


    for(int i=0;i<size;i++){
        dataC[i] = 0;
    }
    hipMemcpy(d_dataA,dataA,sizeof(float) * size,hipMemcpyHostToDevice);
    hipMemcpy(d_dataC,dataC,sizeof(unsigned short int) * size,hipMemcpyHostToDevice);
    test<<<1, 1>>> (d_dataA, d_dataC, size);
    hipMemcpy(dataC,d_dataC,sizeof(unsigned short int) * size, hipMemcpyDeviceToHost);
    std::cout << std::endl;
    
    for(int i=0;i<size;i++){
        FP16 fp16;
        fp16.i = dataC[i];
        fp32.f = dataA[i];
        cout<< "f32 / f16 hex format:,"<<hex<<fp32.i<<"," << dataC[i]<<endl;
        cout<< "f32 / f16 dec format:,"<< std::setprecision(20) <<dataA[i]<<"," <<float(fp16.f)<<endl;
        std::cout << std::endl;
    }

    hipFree(d_dataA);
    hipFree(d_dataC);
    free(dataA);
    free(dataB);
    free(dataC); 
    return 0;
}
