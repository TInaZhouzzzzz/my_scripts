#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<iostream>
#include <iomanip>      // std::setprecision

using namespace std;
union FP32{
    float f;
    unsigned int i;
};

__global__ void test_cvt_f32_to_f16(float* A, short int* C, int N){
    for(int i=0;i<N;i++){
        //asm("cvt.rn.satfinite.e4m3x2.f32 %0, %1, %2;\n\t":"=h"(C[i]):"f"(A[i]),"f"(B[i]));
        asm("cvt.rn.relu.satfinite.f16.f32 %0, %1;\n\t":"=h"(C[i]):"f"(A[i]));
    }
}

__global__ void test_cvt_f32_to_tf32(float* A, unsigned int* C, int N){
    for(int i=0;i<N;i++){
        //asm("cvt.rn.satfinite.e4m3x2.f32 %0, %1, %2;\n\t":"=h"(C[i]):"f"(A[i]),"f"(B[i]));
        asm("cvt.rna.satfinite.tf32.f32 %0, %1;\n\t":"=r"(C[i]):"f"(A[i]));
    }
}

void Initfloat(float * a, const int n) {
  float value;
  for ( int i = 0; i < n; i++ ) {
    value = (float)(rand() % 20 - 10) + (float)(rand() % 20 - 10) / 10.0 + (float)(rand() % 20 - 10) / 100.0 + (float)(rand() % 20 - 10) / 1000.0 + (float)(rand() % 20 - 10) / 10000.0 + (float)(rand() % 20 - 10) / 100000.0 + (float)(rand() % 20 - 10) / 1000000.0 + (float)(rand() % 20 - 10) / 10000000.0 + (float)(rand() % 20 - 10) / 100000000.0;

    a[i] = value;
  }
}

void InitZero(float * a, const int n) {
  for ( int i = 0; i < n; i++ ) {
      a[i] = 0.0;
  }
}


void show(float * a, const int n) {
  std::cout << std::endl;
  for ( int i=0; i<n; i++){ 
    std::cout<< std::setprecision(20) << a[i] << std::endl;
  }
  std::cout << std::endl;
}

int main(int argc, char** argv){
    int size = 10;
    float *dataA = (float*)malloc(sizeof(float) * size);
    unsigned int *dataC = (unsigned int*)malloc(sizeof(unsigned int) * size);
    float *d_dataA = NULL;
    unsigned  int *d_dataC = NULL;
    hipMalloc((void**)&d_dataA, sizeof(float) * size);
    hipMalloc((void**)&d_dataC, sizeof(unsigned int) * size);
    FP32 fp32;

    Initfloat(dataA, size);
    /* Nan */
    fp32.i = 0x7fffffff;    dataA[size-1] = fp32.f;
    fp32.i = 0xffffffff;    dataA[size-2] = fp32.f;

    /* inf */
    fp32.i = 0x7f800000;    dataA[size-3] = fp32.f;
    fp32.i = 0xff800000;    dataA[size-4] = fp32.f;

    /* 0 */
    fp32.i = 0x00000000;    dataA[size-5] = fp32.f;
    fp32.i = 0x80000000;    dataA[size-6] = fp32.f;

    /* overflow */
    fp32.i = 0x7f7ffeba;    dataA[size-7] = fp32.f;
    fp32.i = 0xff7ffeba;    dataA[size-8] = fp32.f;

    /* random normal */
    fp32.i = 0x43acad91;    dataA[size-9] = fp32.f;
    fp32.i = 0xc3acad91;    dataA[size-10] = fp32.f;


//  /* underflow */
//  fp32.i = 0x36801000;    dataA[count] = fp32.f;

//  show(dataA, size);


    for(int i=0;i<size;i++){
        dataC[i] = 0;
    }
    hipMemcpy(d_dataA,dataA,sizeof(float) * size,hipMemcpyHostToDevice);
    hipMemcpy(d_dataC,dataC,sizeof(unsigned int) * size,hipMemcpyHostToDevice);
    test_cvt_f32_to_tf32<<<1, 1>>> (d_dataA, d_dataC, size);
    hipMemcpy(dataC,d_dataC,sizeof(unsigned int) * size, hipMemcpyDeviceToHost);
    std::cout << std::endl;
    
    for(int i=0;i<size;i++){
        FP32 fp32_i;
        FP32 tf32_o;
        fp32_i.f = dataA[i];
        tf32_o.i = dataC[i];
        cout<< "f32 / tf32 hex format:,"<<hex<<fp32_i.i<<"," <<tf32_o.i <<endl;
        cout<< "f32 / tf32 dec format:,"<< std::setprecision(20) <<fp32_i.f<<"," <<tf32_o.f<<endl;
        std::cout << std::endl;
    }

    hipFree(d_dataA);
    hipFree(d_dataC);
    free(dataA);
    free(dataC); 
    return 0;
}
