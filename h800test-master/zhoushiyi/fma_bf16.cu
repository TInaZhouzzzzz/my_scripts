#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<iostream>
#include <iomanip>      // std::setprecision

#include<hip/hip_fp16.h>
#include "include/half.hpp"
using namespace std;
union FP32{
    unsigned short int f;
    unsigned int i;
};

__global__ void test(unsigned short int* A, unsigned short int* B, unsigned short int* C, unsigned short int* D){
    asm("fma.rn.bf16  %0, %1, %2, %3;" : "=h"(D[0]) : "h"(A[0]) , "h"(B[0]) , "h"(C[0]));
}

int main(int argc, char** argv){
    int size = 1;
    unsigned short int *dataA = (unsigned short int*)malloc(sizeof(unsigned short int) * size);
    unsigned short int *dataB = (unsigned short int*)malloc(sizeof(unsigned short int) * size);
    unsigned short int *dataC = (unsigned short int*)malloc(sizeof(unsigned short int) * size);
    unsigned short int *dataD = (unsigned short int*)malloc(sizeof(unsigned short int) * size);
    unsigned short int *d_dataA = NULL;
    unsigned short int *d_dataB = NULL;
    unsigned short int *d_dataC = NULL;
    unsigned short int *d_dataD = NULL;
    hipMalloc((void**)&d_dataA, sizeof(unsigned short int) * size);
    hipMalloc((void**)&d_dataB, sizeof(unsigned short int) * size);
    hipMalloc((void**)&d_dataC, sizeof(unsigned short int) * size);
    hipMalloc((void**)&d_dataD, sizeof(unsigned short int) * size);

    dataA[size-1]  = 0x5d84;
    dataB[size-1]  = 0xa230;
    dataC[size-1]  = 0x0100;


    for(int i=0;i<size;i++){
        dataD[i] = 0;
    }
    hipMemcpy(d_dataA,dataA,sizeof(unsigned short int) * size,hipMemcpyHostToDevice);
    hipMemcpy(d_dataB,dataB,sizeof(unsigned short int) * size,hipMemcpyHostToDevice);
    hipMemcpy(d_dataC,dataC,sizeof(unsigned short int) * size,hipMemcpyHostToDevice);
    test<<<1, 1>>> (d_dataA, d_dataB, d_dataC, d_dataD);
    hipMemcpy(dataD,d_dataD,sizeof(unsigned short int) * size, hipMemcpyDeviceToHost);

    std::cout <<std::hex <<  dataD[size-1] << std::endl;

    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);
    hipFree(d_dataD);
    free(dataA);
    free(dataB);
    free(dataC); 
    free(dataD); 
    return 0;
}
